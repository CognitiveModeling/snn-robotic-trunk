#include "hip/hip_runtime.h"
#ifndef __LONG_SHORT_TERM_MEMORY_FIRING_RATE_KERNEL__
#define __LONG_SHORT_TERM_MEMORY_FIRING_RATE_KERNEL__
#include "CudaUtils.h"
/**
 * Parallel kernel for a Long Short Term Spiking Network
 */
namespace SNN {

    namespace Kernels {

        namespace GPU {

            __global__ void longShortTermMemoryFiringRateKernel(

                /* the number of input neurons */
                unsigned *numInputs_,

                /* the number of (leaky integrate and fire) hiddem neurons */
                unsigned *numStandartHidden_,

                /* the number of (adaptive leaky integrate and fire) hidden neurons */
                unsigned *numAdaptiveHidden_,

                /* the batch size */
                unsigned *batchSize_,

                /* the number of simmulation time steps */
                unsigned *numSimulationTimesteps_,

                /* the simulation timestep length */
                FloatType *timeStepLength_,

                /* the hidden neurons firing rates */
                FloatType *firingRates,

                /* the hidden neurons number of spikes */
                FloatType *numSpikes
            ) {
                const unsigned numInputs              = *numInputs_;
                const unsigned batchSize              = *batchSize_;
                const unsigned numSimulationTimesteps = *numSimulationTimesteps_;
                const FloatType timeStepLength        = *timeStepLength_;

                const unsigned id = threadIdx.x;
                const unsigned numHidden = *numStandartHidden_ + *numAdaptiveHidden_;
                cudaAssert(numHidden == blockDim.x);
                cudaAssert(gridDim.x == 1);
                
                FloatType summedSpikes = 0;
                for (unsigned i = 0; i < batchSize; i++)
                    summedSpikes += numSpikes[i * (numHidden + numInputs) + numInputs + id];

                firingRates[id] = summedSpikes / (batchSize * numSimulationTimesteps * timeStepLength);
            }
        }
    }
}
#endif /* __LONG_SHORT_TERM_MEMORY_FIRING_RATE_KERNEL__ */
