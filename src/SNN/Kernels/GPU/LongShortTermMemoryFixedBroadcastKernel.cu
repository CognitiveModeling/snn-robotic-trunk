#include "hip/hip_runtime.h"
#ifndef __LONG_SHORT_TERM_MEMORY_FIXED_BROADCAST_KERNEL__
#define __LONG_SHORT_TERM_MEMORY_FIXED_BROADCAST_KERNEL__
#include "FullyConnectedLeakyIntegrateAndFireKernel.cu"
#include "FullyConnectedAdaptiveLeakyIntegrateAndFireKernel.cu"
#include "FullyConnectedInputOutputKernel.cu"
#include "FullyConnectedInputSpikePropagationKernel.cu"
#include "FullyConnectedHiddenSpikePropagationKernel.cu"
#include "LongShortTermMemoryLearnSignalKernel.cu"
#include "LongShortTermMemoryEligibilityGradientKernel.cu"
#include "LongShortTermMemoryLeakyReadoutGradientKernel.cu"
#include "LongShortTermMemoryInputEligibilityKernel.cu"
/**
 * Parallel kernel for a Long Short Term Spiking Network
 */
namespace SNN {

    namespace Kernels {

        namespace GPU {

#define ERROR_MODE_REGRESSION 0
#define ERROR_MODE_CLASSIFICATION 1
#define ERROR_MODE_INFINITY 2

            __device__ void longShortTermMemoryFixedBroadcastKernel(

                /* the number of input neurons */
                unsigned numInputs,

                /* the number of (leaky integrate and fire) hiddem neurons */
                unsigned numStandartHidden,

                /* the number of (adaptive leaky integrate and fire) hidden neurons */
                unsigned numAdaptiveHidden,

                /* the number of output neurons */
                unsigned numOutputs,

                /* the number of simmulation time steps */
                unsigned numSimulationTimesteps,

                /* the simulation start and end time */
                int startTime, 
                int endTime,

                /* the error mode of this */
                unsigned errorMode,

                /* the simulation timestep length */
                FloatType timeStepLength,

                /* neuron spike threshold */
                FloatType spikeThreshold,

                /* neuron refactory period */
                FloatType refactoryPeriod,

                /* the hidden voltage decay factor */
                FloatType hiddenDecayFactor,

                /* the readout voltage decay factor */
                FloatType readoutDecayFactor,

                /* the decay factor for the adaptive threshold */
                FloatType adaptationDecayFactor,
                
                /* the factor about which the base threshold increases */
                FloatType thresholdIncreaseConstant,

                /* the target firing rate */
                FloatType targetFiringRate,

                /* the firing rate gradient scalling factor */
                FloatType firingRateScallingFactor,

                /* the derivative dumping factor */
                FloatType derivativeDumpingFactor,

                /* the input neuron spikes over one simulation run */
                FloatType *inputSpikesOverTime,

               /* the input and hidden neuron spikes over one simulation run */
                FloatType *spikesOverTime,

                /* the hidden neurons firing rates */
                FloatType *firingRates,

                /* the hidden neurons number of spikes */
                FloatType *numSpikes,

                /* the synaptic input weights */
                FloatType *inputWeights,

                /* the synaptic input weights */
                FloatType *hiddenWeights,

                /* the synaptic input weights */
                FloatType *outputWeights,

                /* the feedback weights */
                FloatType *feedbackWeights,

                /* the network target weights */
                FloatType *targetWeights,

                /* the network targets fore one simulation run */
                FloatType *targetsOverTime,

                /* the network outputs fore one simulation run */
                FloatType *outputsOverTime,

                /* the network derivatives fore one simulation run */
                FloatType *derivativesOverTime,

                /* the network hidden voltage for the last simulation run */
                FloatType *voltageOverTime,

                /* time since last spike for hidden neurons over time */
                FloatType *timeStepsSinceLastSpikeOverTime,

                /* the neurons adaptation values over time */
                FloatType *thresholdAdaptationOverTime,

                /* the network error mask for one simulation run */
                FloatType *errorMaskOverTime,

                /* the network error factors for one simulation run */
                FloatType *outputErrorFactorOverTime,

                /* the fixed braodcast gradients for input synapses */
                FloatType *inputFixedBroadcastGradients,

                /* the firing rate gradients for input synapses  */
                FloatType *inputFiringRateGradients,

                /* the fixed braodcast gradients for hidden synapses */
                FloatType *hiddenFixedBroadcastGradients,

                /* the firing rate gradients for hidden synapses */
                FloatType *hiddenFiringRateGradients,

                /* the leaky readout gradients */
                FloatType *leakyReadoutGradients,
                
                /* the networks squared summed error */
                FloatType *networkError,

                /* the network summed target for each output */
                FloatType *summedTargets,

                /* the network squared summed target for each output */
                FloatType *squaredSummedTargets,

                /* the number of values summed for each output */
                FloatType *numSummedValues,

                /* the networks classification accuracy error */
                FloatType *classificationAccuracy,

                /* the networks number of classification samples */
                FloatType *classificationSamples,

                /***** content managed by kernel ******/

                /* the filtered eligibility traces */
                FloatType *filteredEligibilityTraces,

                /* the filtered hidden spikes */
                FloatType *filteredSpikes,

                /* the filtered hidden spikes (by the readout decay factor) */
                FloatType *readoutDecayFilteredSpikes,

                /* the neurons adaptation values */
                FloatType *thresholdAdaptation,

                /* the adaption eligibility part */
                FloatType *adaptionEligibility,

                /* the input errors fore one simmulation run */
                FloatType *inputErrorsOverTime,

                /* hidden derivatives */
                FloatType *derivatives,

                /* input current for hidden and output neurons */
                FloatType *I,

                /* hidden and readout voltage */
                FloatType *v,

                /* hidden spikes */
                FloatType *hiddenSpikes,

                /* time since last spike for hidden neurons */
                FloatType *timeStepsSinceLastSpike,

                /* hidden neuron learn signals */
                FloatType *learnSignals,

                /* summed network output for classification */
                FloatType *summedActivation

            ) {
                const unsigned numHidden = numStandartHidden + numAdaptiveHidden;
                cudaAssert(numInputs == blockDim.x || numHidden == blockDim.x);

                inputSpikesOverTime             += blockIdx.x * numInputs * numSimulationTimesteps;
                spikesOverTime                  += blockIdx.x * (numInputs + numHidden) * numSimulationTimesteps;
                numSpikes                       += blockIdx.x * (numHidden + numInputs) + numInputs;
                targetsOverTime                 += blockIdx.x * numOutputs * numSimulationTimesteps;
                outputsOverTime                 += blockIdx.x * numOutputs * numSimulationTimesteps;
                derivativesOverTime             += blockIdx.x * numHidden * numSimulationTimesteps;
                voltageOverTime                 += blockIdx.x * numHidden * numSimulationTimesteps;
                timeStepsSinceLastSpikeOverTime += blockIdx.x * numHidden * numSimulationTimesteps;
                thresholdAdaptationOverTime     += blockIdx.x * numAdaptiveHidden * numSimulationTimesteps;
                errorMaskOverTime               += blockIdx.x * numSimulationTimesteps;
                outputErrorFactorOverTime       += blockIdx.x * numOutputs * numSimulationTimesteps;
                inputFixedBroadcastGradients    += blockIdx.x * numInputs * numHidden;
                inputFiringRateGradients        += blockIdx.x * numInputs * numHidden;
                hiddenFixedBroadcastGradients   += blockIdx.x * numHidden * numHidden;
                hiddenFiringRateGradients       += blockIdx.x * numHidden * numHidden;
                leakyReadoutGradients           += blockIdx.x * numHidden * numOutputs;
                networkError                    += blockIdx.x;
                summedTargets                   += blockIdx.x;
                squaredSummedTargets            += blockIdx.x;
                numSummedValues                 += blockIdx.x;
                filteredEligibilityTraces       += blockIdx.x * (numInputs + numHidden) * numHidden;
                filteredSpikes                  += blockIdx.x * (numInputs + numHidden);
                readoutDecayFilteredSpikes      += blockIdx.x * numHidden;
                thresholdAdaptation             += blockIdx.x * numAdaptiveHidden;
                adaptionEligibility             += blockIdx.x * (numInputs + numHidden) * numAdaptiveHidden;
                inputErrorsOverTime             += blockIdx.x * numInputs * numSimulationTimesteps;
                derivatives                     += blockIdx.x * numHidden;
                I                               += blockIdx.x * (numHidden + numOutputs);
                v                               += blockIdx.x * (numHidden + numOutputs);
                hiddenSpikes                    += blockIdx.x * numHidden;
                timeStepsSinceLastSpike         += blockIdx.x * numHidden;
                learnSignals                    += blockIdx.x * numHidden;
                summedActivation                += blockIdx.x * numOutputs;
                classificationAccuracy          += blockIdx.x;
                classificationSamples           += blockIdx.x;

                if (startTime < 0) startTime = 0;
                if (endTime   < 0) endTime   = numSimulationTimesteps;

                /* clear values */
                const int i = threadIdx.x;
                if (startTime == 0) {
                    if (i < numHidden) {
                        filteredSpikes[i] = 0;
                        readoutDecayFilteredSpikes[i] = 0;
                    }
                }
                if (i < numHidden) {
                    numSpikes[i]      = 0;
                    hiddenSpikes[i]   = 0;
                    I[i]              = 0;
                    v[i]              = 0;
                    timeStepsSinceLastSpike[i] = 2 * refactoryPeriod;
                }

                if (i == 0) {
                    networkError[0] = 0;
                    summedTargets[0] = 0;
                    squaredSummedTargets[0] = 0;
                    numSummedValues[0] = 0;
                    classificationAccuracy[0] = 0;
                    classificationSamples[0] = 0;
                }

                if (startTime == 0 && i < numInputs) {
                    filteredSpikes[i + numHidden] = 0;
                }
                if (i < numOutputs) {
                    I[i + numHidden] = 0;
                    v[i + numHidden] = 0;
                }

                if (i < numAdaptiveHidden)
                    thresholdAdaptation[i] = 0;

                for (unsigned index = 0; index + i < numInputs * numHidden; index += blockDim.x) {
                    inputFiringRateGradients[index + i]     = 0;
                    inputFixedBroadcastGradients[index + i] = 0;
                }
                for (unsigned index = 0; index + i < numHidden * numHidden; index += blockDim.x) {
                    hiddenFiringRateGradients[index + i]     = 0;
                    hiddenFixedBroadcastGradients[index + i] = 0;
                }
                if (startTime == 0) {
                    for (unsigned index = 0; 
                        index + i < (numInputs + numHidden) * numAdaptiveHidden; 
                        index += blockDim.x) {

                        adaptionEligibility[index + i] = 0;
                    }
                    for (unsigned index = 0; 
                        index + i < (numInputs + numHidden) * numHidden; 
                        index += blockDim.x) {

                        filteredEligibilityTraces[index + i] = 0;
                    }
                }
                for (unsigned index = 0; index + i < numHidden * numOutputs; index += blockDim.x) {
                    leakyReadoutGradients[index + i] = 0;
                }

                if (i < numOutputs)
                    summedActivation[i] = 0;

                FloatType adaptiveThreadOffset = 0;
                if (wrapThreads(numStandartHidden) + wrapThreads(numAdaptiveHidden) <= blockDim.x)
                    adaptiveThreadOffset = wrapThreads(numStandartHidden);

                if (startTime > 0) {
                    int lastStart = (startTime - 1) % numSimulationTimesteps;
                    startTime     = startTime       % numSimulationTimesteps;
                    endTime       = ((endTime - 1)  % numSimulationTimesteps) + 1;
                    if (i < numHidden) {
                        hiddenSpikes[i] = spikesOverTime[lastStart * (numInputs + numHidden) + numInputs + i];
                        v[i]            = voltageOverTime[lastStart * numHidden + i];
                        timeStepsSinceLastSpike[i] = timeStepsSinceLastSpikeOverTime[lastStart * numHidden + i];
                    }
                    if (i < numOutputs) 
                        v[numHidden + i] = outputsOverTime[lastStart * numOutputs + i];
                    if (i < numAdaptiveHidden)
                        thresholdAdaptation[i] = thresholdAdaptationOverTime[lastStart * numAdaptiveHidden + i];

                    __syncthreads();
                    fullyConnectedHiddenSpikePropagationKernel(
                        numHidden,
                        numOutputs,
                        hiddenSpikes,
                        I,
                        hiddenWeights,
                        outputWeights
                    );
                }

                __syncthreads();

                for (unsigned t = startTime; t < endTime; t++) {

                    fullyConnectedInputSpikePropagationKernel(
                        numInputs,
                        numHidden,
                        inputSpikesOverTime + t * numInputs,
                        I,
                        inputWeights
                    );
                    __syncthreads();

                    fullyConnectedLeakyIntegrateAndFireKernel(
                        0,
                        numStandartHidden,
                        spikeThreshold,
                        hiddenDecayFactor,
                        refactoryPeriod,
                        derivativeDumpingFactor,
                        hiddenSpikes,
                        filteredSpikes + numInputs,
                        numSpikes,
                        I,
                        v,
                        derivatives,
                        timeStepsSinceLastSpike
                    );
                    fullyConnectedAdaptiveLeakyIntegrateAndFireKernel(
                        adaptiveThreadOffset,
                        numAdaptiveHidden,
                        spikeThreshold,
                        hiddenDecayFactor,
                        adaptationDecayFactor,
                        thresholdIncreaseConstant,
                        refactoryPeriod,
                        derivativeDumpingFactor,
                        hiddenSpikes + numStandartHidden,
                        thresholdAdaptation,
                        filteredSpikes + numInputs + numStandartHidden,
                        numSpikes + numStandartHidden,
                        I + numStandartHidden,
                        v + numStandartHidden,
                        derivatives + numStandartHidden,
                        timeStepsSinceLastSpike + numStandartHidden
                    );

                    __syncthreads();

                    fullyConnectedHiddenSpikePropagationKernel(
                        numHidden,
                        numOutputs,
                        hiddenSpikes,
                        I,
                        hiddenWeights,
                        outputWeights
                    );

                    __syncthreads();

                    fullyConnectedInputOutputKernel(
                        numInputs, 
                        numOutputs,
                        hiddenDecayFactor,
                        readoutDecayFactor,
                        I + numHidden, 
                        v + numHidden, 
                        inputSpikesOverTime + t * numInputs,
                        filteredSpikes
                    );

                    __syncthreads();

                    if (i < numOutputs)
                        outputsOverTime[t * numOutputs + i] = v[numHidden + i];

                    if (i < numHidden) {
                        derivativesOverTime[t * numHidden + i] = derivatives[i];
                        voltageOverTime[t * numHidden + i] = v[i];
                        spikesOverTime[t * (numInputs + numHidden) + numInputs + i] = hiddenSpikes[i];
                        timeStepsSinceLastSpikeOverTime[t * numHidden + i] = timeStepsSinceLastSpike[i];
                    }

                    if (i < numAdaptiveHidden)
                        thresholdAdaptationOverTime[t * numAdaptiveHidden + i] = thresholdAdaptation[i];

                    if (i < numInputs)
                        spikesOverTime[t * (numInputs + numHidden) + i] = inputSpikesOverTime[t * numInputs + i];

                    if (i < numHidden)
                        spikesOverTime[t * (numInputs + numHidden) + numInputs + i] = hiddenSpikes[i];

                    if (i == 0 && errorMaskOverTime[t] != 0) {
                        for (unsigned o = 0; o < numOutputs; o++) {
                            summedTargets[0] += targetsOverTime[t * numOutputs + o];
                            squaredSummedTargets[0] += pow(targetsOverTime[t * numOutputs + o], 2);
                            numSummedValues[0] += 1;
                        }
                        if (errorMode == ERROR_MODE_INFINITY) {
                            for (unsigned o = 0; o < numOutputs; o++) {
                                networkError[0] += exp(
                                    -1.0 * targetsOverTime[t * numOutputs + o] * v[numHidden + o]
                                );
                            }

                            for (unsigned o = 0; o < numOutputs; o++) 
                                summedActivation[o] += v[numHidden + o];

                            if (t + 1 == numSimulationTimesteps || errorMaskOverTime[t + 1] == 0) {

                                for (unsigned o = 0; o < numOutputs; o++) {
                                    const FloatType target = targetsOverTime[t * numOutputs + o];

                                    classificationSamples[0] += fabs(target);
                                    if (summedActivation[o] * target > 0)
                                        classificationAccuracy[0] += fabs(target);
                                }
                                    
                                for (unsigned o = 0; o < numOutputs; o++) 
                                    summedActivation[o] = 0;
                            }
                        } else if (errorMode == ERROR_MODE_CLASSIFICATION) {
                            FloatType expSum = 0;
                            for (unsigned o = 0; o < numOutputs; o++) 
                                expSum += exp(v[numHidden + o]);

                            for (unsigned o = 0; o < numOutputs; o++) {
                                const FloatType softmax = exp(v[numHidden + o]) / expSum;
                                networkError[0] -= targetsOverTime[t * numOutputs + o] *
                                                   log(softmax);
                            }

                            for (unsigned o = 0; o < numOutputs; o++) 
                                summedActivation[o] += v[numHidden + o];

                            if (t + 1 == numSimulationTimesteps || errorMaskOverTime[t + 1] == 0) {
                                unsigned maxNeuron = 0;

                                for (unsigned o = 1; o < numOutputs; o++) 
                                    if (summedActivation[o] > summedActivation[maxNeuron])
                                        maxNeuron = o;
                                    
                                classificationAccuracy[0] += targetsOverTime[t * numOutputs + maxNeuron];
                                classificationSamples[0]++;

                                for (unsigned o = 0; o < numOutputs; o++) 
                                    summedActivation[o] = 0;
                            }
                        } else if (errorMode == ERROR_MODE_REGRESSION) {
                            for (unsigned o = 0; o < numOutputs; o++) {
                                networkError[0] += pow(
                                    v[numHidden + o] - 
                                    targetsOverTime[t * numOutputs + o], 
                                    2
                                );
                            }
                        }
                    }

                    longShortTermMemoryLearnSignalKernel(
                        numHidden,
                        numOutputs,
                        errorMode,
                        v + numHidden,
                        targetWeights,
                        targetsOverTime + t * numOutputs,
                        errorMaskOverTime[t],
                        outputErrorFactorOverTime + t * numOutputs,
                        learnSignals,
                        feedbackWeights
                    );

                    __syncthreads();

                    longShortTermMemoryEligibilityGradientKernel(
                        numInputs,
                        numStandartHidden,
                        numAdaptiveHidden,
                        targetFiringRate,
                        firingRateScallingFactor,
                        readoutDecayFactor,
                        adaptationDecayFactor,
                        thresholdIncreaseConstant,
                        filteredSpikes,
                        firingRates,
                        derivatives,
                        learnSignals,
                        adaptionEligibility,
                        filteredEligibilityTraces,
                        inputFiringRateGradients,
                        inputFixedBroadcastGradients
                    );
                    longShortTermMemoryEligibilityGradientKernel(
                        numHidden,
                        numStandartHidden,
                        numAdaptiveHidden,
                        targetFiringRate,
                        firingRateScallingFactor,
                        readoutDecayFactor,
                        adaptationDecayFactor,
                        thresholdIncreaseConstant,
                        filteredSpikes + numInputs,
                        firingRates,
                        derivatives,
                        learnSignals,
                        adaptionEligibility + numInputs * numAdaptiveHidden,
                        filteredEligibilityTraces + numInputs * numHidden,
                        hiddenFiringRateGradients,
                        hiddenFixedBroadcastGradients
                    );
                    longShortTermMemoryLeakyReadoutGradientKernel(
                        numHidden,
                        numOutputs,
                        errorMode,
                        readoutDecayFactor,
                        readoutDecayFilteredSpikes, 
                        hiddenSpikes,
                        v + numHidden,
                        targetsOverTime + t * numOutputs,
                        errorMaskOverTime[t],
                        leakyReadoutGradients
                    );

                    __syncthreads();
                }
            }
        }
    }
}
#endif /* __LONG_SHORT_TERM_MEMORY_FIXED_BROADCAST_KERNEL__ */
