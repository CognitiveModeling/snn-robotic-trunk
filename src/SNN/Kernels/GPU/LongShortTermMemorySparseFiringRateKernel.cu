#include "hip/hip_runtime.h"
#ifndef __LONG_SHORT_TERM_MEMORY_SPARSE_FIRING_RATE_KERNEL__
#define __LONG_SHORT_TERM_MEMORY_SPARSE_FIRING_RATE_KERNEL__
#include "CudaUtils.h"
/**
 * Parallel kernel for a Long Short Term Spiking Network
 */
namespace SNN {

    namespace Kernels {

        namespace GPU {

            __global__ void longShortTermMemorySparseFiringRateKernel(

                /* the number of input neurons */
                unsigned *numInputs_,

                /* the number of (leaky integrate and fire) hiddem neurons */
                unsigned *numStandartHidden_,

                /* the number of (adaptive leaky integrate and fire) hidden neurons */
                unsigned *numAdaptiveHidden_,

                /* the batch size */
                unsigned *batchSize_,

                /* the number of simmulation time steps */
                unsigned *numSimulationTimesteps_,

                /* the simulation timestep length */
                FloatType *timeStepLength_,

                /* the hidden neurons firing rates */
                FloatType *firingRates,

                /* the hidden neurons number of spikes */
                FloatType *numSpikes
            ) {
                const unsigned numInputs              = *numInputs_;
                const unsigned batchSize              = *batchSize_;
                const unsigned numSimulationTimesteps = *numSimulationTimesteps_;
                const FloatType timeStepLength        = *timeStepLength_;

                const unsigned numHidden = *numStandartHidden_ + *numAdaptiveHidden_;
                
                for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; 
                     i < numHidden;
                     i += gridDim.x * blockDim.x) {

                    FloatType summedSpikes = 0;
                    for (unsigned b = 0; b < batchSize; b++)
                        summedSpikes += numSpikes[b * (numInputs + numHidden) + numInputs + i];

                    firingRates[i] = summedSpikes / (batchSize * numSimulationTimesteps * timeStepLength);
                }
            }
        }
    }
}
#endif /* __LONG_SHORT_TERM_MEMORY_SPARSE_FIRING_RATE_KERNEL__ */
